#include "hip/hip_runtime.h"
#include <stdio.h>

/** CUDA Back-End **/
#include "mm_cuda_backend.cuh"

// non-specialized class template
template<typename T>
class SharedMem {
public:
	// Ensure that we won't compile any un-specialized types
	__device__
	T* getPointer() {
		return (T*) NULL;
	}
	;
};

// specialization for double
template<>
class SharedMem<double> {
public:
	__device__
	double* getPointer() {
		extern __shared__ double s_db[]; return s_db;
	}
};

// specialization for float
template<>
class SharedMem<float> {
public:
	__device__
	float* getPointer() {
		extern __shared__ float s_fl[]; return s_fl;
	}
};

// specialization for unsigned long long
template<>
class SharedMem<unsigned long long> {
public:
	__device__
	unsigned long long* getPointer() {
		extern __shared__ unsigned long long s_ul[]; return s_ul;
	}
};

// specialization for int
template<>
class SharedMem<int> {
public:
	__device__
	int* getPointer() {
		extern __shared__ int s_in[]; return s_in;
	}
};

// specialization for unsigned int
template<>
class SharedMem<unsigned int> {
public:
	__device__
	unsigned int* getPointer() {
		extern __shared__ unsigned int s_ui[]; return s_ui;
	}
};

template <typename T>
__device__ void block_reduction(T *psum, int tid, int len_) {
	int stride = len_ >> 1;
	while (stride > 0) {
		//while (stride > 32) {
		if (tid < stride) psum[tid] += psum[tid+stride];
		__syncthreads();
		stride >>= 1;
	}
	__syncthreads();
	/*if (tid < 32) { 
	 psum[tid] += psum[tid+32];
	 __syncthreads();
	 psum[tid] += psum[tid+16];
	 __syncthreads();
	 psum[tid] += psum[tid+8];
	 __syncthreads();
	 psum[tid] += psum[tid+4];
	 __syncthreads();
	 psum[tid] += psum[tid+2];
	 __syncthreads();
	 psum[tid] += psum[tid+1];
	 __syncthreads();
	 }*/
}

//TODO figure out how to use templates with the __X_as_Y intrinsics
//Paul - Implementation of double atomicAdd from CUDA Programming Guide: Appendix B.12
__device__ double atomicAdd(double* address, double val) {
	unsigned long long int* address_as_ull = (unsigned long long int*) address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
				__double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}

//wrapper for all the types natively supported by CUDA
template <typename T> __device__ T atomicAdd_wrapper(T* addr, T val) {
	return atomicAdd(addr, val);
}

//wrapper for had-implemented double type
template <> __device__ double atomicAdd_wrapper<double>(double* addr, double val) {
	return atomicAdd(addr, val);
}

// this kernel works for 3D data only.
//  PHI1 and PHI2 are input arrays.
//  s* parameters are start values in each dimension.
//  e* parameters are end values in each dimension.
//  s* and e* are only necessary when the halo layers 
//    has different thickness along various directions.
//  i,j,k are the array dimensions
//  len_ is number of threads in a threadblock.
//       This can be computed in the kernel itself.
template <typename T>
__global__ void kernel_dotProd(T *phi1, T *phi2,
		int i, int j, int k,
		int sx, int sy, int sz,
		int ex, int ey, int ez,
		int gz, T * reduction, int len_) {
	SharedMem<T> shared;
	T * psum = shared.getPointer();
	int tid, loads, x, y, z, itr;
	bool boundx, boundy, boundz;
	tid = threadIdx.x+(threadIdx.y)*blockDim.x+(threadIdx.z)*(blockDim.x*blockDim.y);

	x = (blockIdx.x)*blockDim.x+threadIdx.x+sx;
	y = (blockIdx.y)*blockDim.y+threadIdx.y+sy;

	loads = gz;

	psum[tid] = 0;
	boundy = ((y >= sy) && (y <= ey));
	boundx = ((x >= sx) && (x <= ex));

	for (itr = 0; itr < loads; itr++) {
		z = itr*blockDim.z+threadIdx.z +sz;
		boundz = ((z >= sz) && (z <= ez));
		//if (boundx && boundy && boundz) psum[tid] += phi1[x+y*i+z*i*j] * phi2[x+y*i+z*i*j];
		if (boundx && boundy && boundz) psum[tid] += phi1[x+y*i+z*i*j] * phi2[x+y*i+z*i*j];
	}

	__syncthreads();
	//After accumulating the Z-dimension, have each block internally reduce X and Y
	block_reduction<T>(psum,tid,len_);
	__syncthreads();

	//Merge reduced values from all blocks
	if(tid == 0) atomicAdd_wrapper<T>(reduction,psum[0]);
}

template <typename T>
__global__ void kernel_reduction3(T *phi,
		int i, int j, int k,
		int sx, int sy, int sz,
		int ex, int ey, int ez,
		int gz, T * reduction, int len_) {
	SharedMem<T> shared;
	T *psum = shared.getPointer();
	int tid, loads, x, y, z, itr;
	bool boundx, boundy, boundz;
	tid = threadIdx.x+(threadIdx.y)*blockDim.x+(threadIdx.z)*(blockDim.x*blockDim.y);

	x = (blockIdx.x)*blockDim.x+threadIdx.x+sx;
	y = (blockIdx.y)*blockDim.y+threadIdx.y+sy;

	loads = gz;

	psum[tid] = 0;
	boundy = ((y >= sy) && (y <= ey));
	boundx = ((x >= sx) && (x <= ex));

	for (itr = 0; itr < loads; itr++) {
		z = itr*blockDim.z+threadIdx.z +sz;
		boundz = ((z >= sz) && (z <= ez));
		if (boundx && boundy && boundz) psum[tid] += phi[x+y*i+z*i*j];
	}

	__syncthreads();
	//After accumulating the Z-dimension, have each block internally reduce X and Y
	block_reduction<T>(psum,tid,len_);
	__syncthreads();

	//Merge reduced values from all blocks
	if(tid == 0) atomicAdd_wrapper<T>(reduction,psum[0]);
}

//"constant" buffers for face indexing in pack/unpack kernels
__constant__ int c_face_size[METAMORPH_FACE_MAX_DEPTH];
__constant__ int c_face_stride[METAMORPH_FACE_MAX_DEPTH];
//Size of all children (>= level+1) so at level 0, child_size = total_num_face_elements
__constant__ int c_face_child_size[METAMORPH_FACE_MAX_DEPTH];

//Helper function to compute the integer read offset for buffer packing
//TODO: Add support for multi-dimensional grid/block
__device__ int get_pack_index(int tid, int * a, int start, int count) {
	int pos;
	int i, j, k, l;
	for (i = 0; i < count; i++)
		a[tid % blockDim.x + i * blockDim.x] = 0;

	for (i = 0; i < count; i++) {
		k = 0;
		for (j = 0; j < i; j++) {
			k += a[tid % blockDim.x + j * blockDim.x] * c_face_child_size[j];
		}
		l = c_face_child_size[i];
		for (j = 0; j < c_face_size[i]; j++) {
			if (tid - k < l)
				break;
			else
				l += c_face_child_size[i];
		}
		a[tid % blockDim.x + i * blockDim.x] = j;
	}
	pos = start;
	for (i = 0; i < count; i++) {
		pos += a[tid % blockDim.x + i * blockDim.x] * c_face_stride[i];
	}
	return pos;
}

template <typename T>
__global__ void kernel_pack(T *packed_buf, T *buf, int size, int start, int count)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int nthreads = gridDim.x * blockDim.x;
	extern __shared__ int a[];
	// this loop handles both nthreads > size and nthreads < size
	for (; idx < size; idx += nthreads)
	packed_buf[idx] = buf[get_pack_index(idx, a, start, count)];
}

template <typename T>
__global__ void kernel_unpack(T *packed_buf, T *buf, int size, int start, int count)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int nthreads = gridDim.x * blockDim.x;
	extern __shared__ int a[];
	// this loop handles both nthreads > size and nthreads < size
	for (; idx < size; idx += nthreads)
	buf[get_pack_index(idx, a, start, count)] = packed_buf[idx];
}

//TODO: Expand to multiple transpose elements per thread
//#define TRANSPOSE_TILE_DIM (16)
//#define TRANSPOSE_BLOCK_ROWS (16)
template <typename T>
__global__ void kernel_transpose_2d(T *odata, T *idata, int arr_width, int arr_height, int tran_width, int tran_height)
{
	SharedMem<T> shared;
	T * tile = shared.getPointer();
	//__shared__ T tile[TRANSPOSE_TILE_DIM][TRANSPOSE_TILE_DIM+1];
	//__shared__ T tile[TRANSPOSE_TILE_DIM][TRANSPOSE_TILE_DIM];

	int blockIdx_x, blockIdx_y;
	int gridDim_x, gridDim_y;

// do diagonal reordering
	//The if case degenerates to the else case, no need to have both
	//if (width == height)
	//{
	//    blockIdx_y = blockIdx.x;
	//    blockIdx_x = (blockIdx.x+blockIdx.y)%gridDim.x;
	//}
	//else
	//{
	//First figure out your number among the actual grid blocks
	int bid = blockIdx.x + gridDim.x*blockIdx.y;
	//Then figure out how many logical blocks are required in each dimension
	gridDim_x = (tran_width-1+blockDim.x)/blockDim.x;
	gridDim_y = (tran_height-1+blockDim.y)/blockDim.y;
	//Then how many logical and actual grid blocks
	int logicalBlocks = gridDim_x*gridDim_y;
	int gridBlocks = gridDim.x*gridDim.y;
	//Loop over all logical blocks
	for (; bid < logicalBlocks; bid += gridBlocks) {
		//Compute the current logical block index in each dimension
		blockIdx_y = bid%gridDim_y;
		blockIdx_x = ((bid/gridDim_y)+blockIdx_y)%gridDim_x;
		//}

		//int xIndex_in = blockIdx_x * TRANSPOSE_TILE_DIM + threadIdx.x;
		int xIndex_in = blockIdx_x * blockDim.x + threadIdx.x;
		//int yIndex_in = blockIdx_y * TRANSPOSE_TILE_DIM + threadIdx.y;
		int yIndex_in = blockIdx_y * blockDim.y + threadIdx.y;
		//int index_in = xIndex_in + (yIndex_in)*width;
		int index_in = xIndex_in + (yIndex_in)*arr_width;

		//int xIndex_out = blockIdx_y * TRANSPOSE_TILE_DIM + threadIdx.x;
		int xIndex_out = blockIdx_y * blockDim.y + threadIdx.x;
		//int yIndex_out = blockIdx_x * TRANSPOSE_TILE_DIM + threadIdx.y;
		int yIndex_out = blockIdx_x * blockDim.x + threadIdx.y;
		//int index_out = xIndex_out + (yIndex_out)*height;
		int index_out = xIndex_out + (yIndex_out)*arr_height;

		//The blockDim.x+1 in R/W steps is to avoid bank conflicts if blockDim.x==16 or 32
		if(xIndex_in < tran_width && yIndex_in < tran_height)
		//tile[threadIdx.y+1][threadIdx.x] = idata[index_in];
		tile[threadIdx.x+(blockDim.x+1)*threadIdx.y] = idata[index_in];
		//tile[threadIdx.y][threadIdx.x] = idata[index_in];

		__syncthreads();

		if(xIndex_out < tran_height && yIndex_out < tran_width)
		//if(xIndex_out < tran_width && yIndex_out < tran_height)
		//odata[index_out] = tile[threadIdx.x][threadIdx.y];
		odata[index_out] = tile[threadIdx.y+(blockDim.y+1)*threadIdx.x];
		//odata[index_out] = tile[threadIdx.x][threadIdx.y];

		//Added when the loop was added to ensure writes are finished before new vals go into shared memory
		__syncthreads();
	}
}

// this kernel works for 3D data only.
//  i,j,k are the array dimensions
//  s* parameters are start values in each dimension.
//  e* parameters are end values in each dimension.
//  s* and e* are only necessary when the halo layers
//    has different thickness along various directions.
//  len_ is number of threads in a threadblock.
//       This can be computed in the kernel itself.
template <typename T>

//Read-only cache + Rigster blocking (Z) + smem blocking (X-Y)
// work only with 2D thread blocks (use rectangular blocks, i.e. 64*4, 128*2)
__global__ void kernel_stencil_3d7p(const T * __restrict__ ind, T * __restrict__ outd,
		int i, int j, int k,
		int sx, int sy, int sz,
		int ex, int ey, int ez,
		int gz, int len_) {
	SharedMem<T> shared;
	T * bind = shared.getPointer(); //[blockDim.x+2*blockDim.y+2]
	const int bi = (blockDim.x+2);
	const int bc = (threadIdx.x+1)+(threadIdx.y+1)*bi;
	T r0, rz1, rz2;
	int x, y, z;
	int ij = i*j;
	int c;
	bool boundx, boundy, boundz;

	x = (blockIdx.x)*blockDim.x+threadIdx.x+sx;
	y = (blockIdx.y)*blockDim.y+threadIdx.y+sy;
	z = threadIdx.z +sz;//blockDim.z ==1
	c = x+y*i+z*ij;
	r0 = ind[c];
	rz1 = ind[c-ij];
	rz2 = ind[c+ij];

	boundy = ((y > sy) && (y < ey));
	boundx = ((x > sx) && (x < ex));
#pragma unroll 8
	for (; z < gz; z++) {
		boundz = ((z > sz) && (z < ez));
		bind[bc] = r0;

		if(threadIdx.x == 0)
		bind[bc-1] = ind[c-1];
		else if (threadIdx.x == blockDim.x-1)
		bind[bc+1] = ind[c+1];

		if(threadIdx.y == 0)
		bind[bc-bi] = ind[c-i];
		else if (threadIdx.y == blockDim.y-1)
		bind[bc+bi] = ind[c+i];

		__syncthreads();

		if (boundx && boundy && boundz)
		outd[c] = ( rz1 + bind[bc-1] + bind[bc-bi] + r0 +
				bind[bc+bi] + bind[bc+1] + rz2 ) / (T) 7;
		c += ij;
		rz1 = r0;
		r0 = rz2;
		rz2 = ind[c+ij];

		__syncthreads();
	}
}

#if 0
template <typename T>
// work with 2D and 3D thread blocks
__global__ void kernel_stencil_3d7p_v0(T *ind, T *outd,
		int i, int j, int k,
		int sx, int sy, int sz,
		int ex, int ey, int ez,
		int gz, int len_) {
	int x, y, z, itr;
	bool boundx, boundy, boundz;

	x = (blockIdx.x)*blockDim.x+threadIdx.x+sx;
	y = (blockIdx.y)*blockDim.y+threadIdx.y+sy;

	boundy = ((y > sy) && (y < ey));
	boundx = ((x > sx) && (x < ex));

	for (itr = 0; itr < gz; itr++) {
		z = itr*blockDim.z+threadIdx.z +sz;
		boundz = ((z > sz) && (z < ez));
		if (boundx && boundy && boundz)
		outd[x+y*i+z*i*j] = ( ind[x+y*i+(z-1)*i*j] + ind[(x-1)+y*i+z*i*j] + ind[x+(y-1)*i+z*i*j] +
				ind[x+y*i+z*i*j] + ind[x+(y+1)*i+z*i*j] + ind[(x+1)+y*i+z*i*j] +
				ind[x+y*i+(z+1)*i*j] ) / (T) 7;
	}
}

template <typename T>
//Read-only cache + Rigster blocking (Z)
// work only with 2D thread blocks
__global__ void kernel_stencil_3d7p_v1(const T * __restrict__ ind, T * __restrict__ outd,
		int i, int j, int k,
		int sx, int sy, int sz,
		int ex, int ey, int ez,
		int gz, int len_) {
	T r0, rz1, rz2;
	int x, y, z;
	int ij = i*j;
	int c;
	bool boundx, boundy, boundz;

	x = (blockIdx.x)*blockDim.x+threadIdx.x+sx;
	y = (blockIdx.y)*blockDim.y+threadIdx.y+sy;
	z = threadIdx.z +sz; //blockDim.z ==1
	c = x+y*i+z*ij;
	r0 = ind[c];
	rz1 = ind[c-ij];
	rz2 = ind[c+ij];

	boundy = ((y > sy) && (y < ey));
	boundx = ((x > sx) && (x < ex));
#pragma unroll 8
	for (; z < gz; z++) {
		boundz = ((z > sz) && (z < ez));
		if (boundx && boundy && boundz)
		outd[c] = ( rz1 + ind[c-1] + ind[c-i] + r0 +
				ind[c+i] + ind[c+1] + rz2 ) / (T) 7;
		c += ij;
		rz1 = r0;
		r0 = rz2;
		rz2 = ind[c+ij];
	}
}

template <typename T>
//Read-only cache + Rigster blocking (Z) + manual prefetch
// work only with 2D thread blocks
__global__ void kernel_stencil_3d7p_v2(const T * __restrict__ ind, T * __restrict__ outd,
		int i, int j, int k,
		int sx, int sy, int sz,
		int ex, int ey, int ez,
		int gz, int len_) {
	T r0, rz1, rz2, rz3;
	int x, y, z;
	int ij = i*j;
	int c;
	bool boundx, boundy, boundz;

	x = (blockIdx.x)*blockDim.x+threadIdx.x+sx;
	y = (blockIdx.y)*blockDim.y+threadIdx.y+sy;
	z = threadIdx.z +sz; //blockDim.z ==1
	c = x+y*i+z*ij;
	r0 = ind[c];
	rz1 = ind[c-ij];
	rz2 = ind[c+ij];
	rz3 = ind[c+ij*2];

	boundy = ((y > sy) && (y < ey));
	boundx = ((x > sx) && (x < ex));
#pragma unroll 8
	for (; z < gz; z++) {
		boundz = ((z > sz) && (z < ez));
		if (boundx && boundy && boundz)
		outd[c] = ( rz1 + ind[c-1] + ind[c-i] + r0 +
				ind[c+i] + ind[c+1] + rz2 ) / (T) 7;
		c += ij;
		rz1 = r0;
		r0 = rz2;
		rz2 = rz3;
		rz3 = ind[c+ij*2];
	}
}

template <typename T>

//Read-only cache + Rigster blocking (Z) + + smem blocking (X-Y)
// work only with 2D thread blocks
__global__ void kernel_stencil_3d7p_v3(const T * __restrict__ ind, T * __restrict__ outd,
		int i, int j, int k,
		int sx, int sy, int sz,
		int ex, int ey, int ez,
		int gz, int len_) {
	SharedMem<T> shared;
	T * bind = shared.getPointer(); //[blockDim.x+2*blockDim.y+2]
	const int bi = (blockDim.x+2);
	const int bc = (threadIdx.x+1)+(threadIdx.y+1)*bi;
	T r0, rz1, rz2;
	int x, y, z;
	int ij = i*j;
	int c;
	bool boundx, boundy, boundz;

	x = (blockIdx.x)*blockDim.x+threadIdx.x+sx;
	y = (blockIdx.y)*blockDim.y+threadIdx.y+sy;
	z = threadIdx.z +sz;//blockDim.z ==1
	c = x+y*i+z*ij;
	r0 = ind[c];
	rz1 = ind[c-ij];
	rz2 = ind[c+ij];

	boundy = ((y > sy) && (y < ey));
	boundx = ((x > sx) && (x < ex));
#pragma unroll 8
	for (; z < gz; z++) {
		boundz = ((z > sz) && (z < ez));
		bind[bc] = r0;

		if(threadIdx.x == 0)
		bind[bc-1] = ind[c-1];
		else if (threadIdx.x == blockDim.x-1)
		bind[bc+1] = ind[c+1];

		if(threadIdx.y == 0)
		bind[bc-bi] = ind[c-i];
		else if (threadIdx.y == blockDim.y-1)
		bind[bc+bi] = ind[c+i];

		__syncthreads();

		if (boundx && boundy && boundz)
		outd[c] = ( rz1 + bind[bc-1] + bind[bc-bi] + r0 +
				bind[bc+bi] + bind[bc+1] + rz2 ) / (T) 7;
		c += ij;
		rz1 = r0;
		r0 = rz2;
		rz2 = ind[c+ij];
	}
}

template <typename T>

// explicit Read-only cache + Rigster blocking (Z) + + smem blocking (X-Y)
// work only with 2D thread blocks
__global__ void kernel_stencil_3d7p_v4(const T * __restrict__ ind, T * __restrict__ outd,
		int i, int j, int k,
		int sx, int sy, int sz,
		int ex, int ey, int ez,
		int gz, int len_) {
	SharedMem<T> shared;
	T * bind = shared.getPointer(); //[blockDim.x+2*blockDim.y+2]
	const int bi = (blockDim.x+2);
	const int bc = (threadIdx.x+1)+(threadIdx.y+1)*bi;
	T r0, rz1, rz2;
	int x, y, z;
	int ij = i*j;
	int c;
	bool boundx, boundy, boundz;

	x = (blockIdx.x)*blockDim.x+threadIdx.x+sx;
	y = (blockIdx.y)*blockDim.y+threadIdx.y+sy;
	z = threadIdx.z +sz;//blockDim.z ==1
	c = x+y*i+z*ij;
	r0 = __ldg(&ind[c]);
	rz1 = __ldg(&ind[c-ij]);
	rz2 = __ldg(&ind[c+ij]);

	boundy = ((y > sy) && (y < ey));
	boundx = ((x > sx) && (x < ex));
#pragma unroll 8
	for (; z < gz; z++) {
		boundz = ((z > sz) && (z < ez));
		bind[bc] = r0;

		if(threadIdx.x == 0)
		bind[bc-1] = __ldg(&ind[c-1]);
		else if (threadIdx.x == blockDim.x-1)
		bind[bc+1] = __ldg(&ind[c+1]);

		if(threadIdx.y == 0)
		bind[bc-bi] = __ldg(&ind[c-i]);
		else if (threadIdx.y == blockDim.y-1)
		bind[bc+bi] = __ldg(&ind[c+i]);

		__syncthreads();

		if (boundx && boundy && boundz)
		outd[c] = ( rz1 + bind[bc-1] + bind[bc-bi] + r0 +
				bind[bc+bi] + bind[bc+1] + rz2 ) / (T) 7;
		c += ij;
		rz1 = r0;
		r0 = rz2;
		rz2 = __ldg(&ind[c+ij]);
	}
}
#endif
/// END KERNELS

/// BEGIN HOST WRAPPERS

hipError_t cuda_dotProd(size_t (*grid_size)[3], size_t (*block_size)[3],
		void * data1, void * data2, size_t (*array_size)[3],
		size_t (*arr_start)[3], size_t (*arr_end)[3], void * reduced_val,
		meta_type_id type, int async, hipEvent_t ((*event)[2])) {
	hipError_t ret = hipSuccess;
	size_t smem_len;
	dim3 grid;
	dim3 block;
	int iters;
	//Allow for auto-selected grid/block size if either is not specified
	if (grid_size == NULL || block_size == NULL) {
		block = METAMORPH_CUDA_DEFAULT_BLOCK;
		//do not subtract 1 from blocx for the case when end == start
		grid = dim3((((*arr_end)[0] - (*arr_start)[0] + (block.x)) / block.x),
				(((*arr_end)[1] - (*arr_start)[1] + (block.y)) / block.y), 1);
		iters = (((*arr_end)[2] - (*arr_start)[2] + (block.z)) / block.z);
	} else {
		grid = dim3((*grid_size)[0], (*grid_size)[1], 1);
		block = dim3((*block_size)[0], (*block_size)[1], (*block_size)[2]);
		iters = (int) (*grid_size)[2];
	}
	smem_len = block.x * block.y * block.z;
	if (event != NULL) {
		hipEventCreate(&(*event)[0]);
		hipEventRecord((*event)[0], 0);
	}
	switch (type) {
	case a_db:
		kernel_dotProd<double><<<grid,block,smem_len*sizeof(double)>>>((double*)data1, (double*)data2, (*array_size)[0], (*array_size)[1], (*array_size)[2], (*arr_start)[0], (*arr_start)[1], (*arr_start)[2], (*arr_end)[0], (*arr_end)[1], (*arr_end)[2], iters, (double*)reduced_val, smem_len);
		break;

	case a_fl:
		kernel_dotProd<float><<<grid,block,smem_len*sizeof(float)>>>((float*)data1, (float*)data2, (*array_size)[0], (*array_size)[1], (*array_size)[2], (*arr_start)[0], (*arr_start)[1], (*arr_start)[2], (*arr_end)[0], (*arr_end)[1], (*arr_end)[2], iters, (float*)reduced_val, smem_len);
		break;

	case a_ul:
		kernel_dotProd<unsigned long long><<<grid,block,smem_len*sizeof(unsigned long long)>>>((unsigned long long*)data1, (unsigned long long*)data2, (*array_size)[0], (*array_size)[1], (*array_size)[2], (*arr_start)[0], (*arr_start)[1], (*arr_start)[2], (*arr_end)[0], (*arr_end)[1], (*arr_end)[2], iters, (unsigned long long*)reduced_val, smem_len);
		break;

	case a_in:
		kernel_dotProd<int><<<grid,block,smem_len*sizeof(int)>>>((int*)data1, (int*)data2, (*array_size)[0], (*array_size)[1], (*array_size)[2], (*arr_start)[0], (*arr_start)[1], (*arr_start)[2], (*arr_end)[0], (*arr_end)[1], (*arr_end)[2], iters, (int *)reduced_val, smem_len);
		break;

	case a_ui:
		kernel_dotProd<unsigned int><<<grid,block,smem_len*sizeof(unsigned int)>>>((unsigned int*)data1, (unsigned int*)data2, (*array_size)[0], (*array_size)[1], (*array_size)[2], (*arr_start)[0], (*arr_start)[1], (*arr_start)[2], (*arr_end)[0], (*arr_end)[1], (*arr_end)[2], iters, (unsigned int*)reduced_val, smem_len);
		break;

	default:
		fprintf(stderr,
				"Error: Function 'cuda_dotProd' not implemented for selected type!\n");
		break;
	}
	if (event != NULL) {
		hipEventCreate(&(*event)[1]);
		hipEventRecord((*event)[1], 0);
	}
	if (!async)
		ret = hipDeviceSynchronize();
	return (ret);
}

hipError_t cuda_reduce(size_t (*grid_size)[3], size_t (*block_size)[3],
		void * data, size_t (*array_size)[3], size_t (*arr_start)[3],
		size_t (*arr_end)[3], void * reduced_val, meta_type_id type, int async,
		hipEvent_t ((*event)[2])) {
	hipError_t ret = hipSuccess;
	size_t smem_len;
	dim3 grid;
	dim3 block;
	int iters;
	//Allow for auto-selected grid/block size if either is not specified
	if (grid_size == NULL || block_size == NULL) {
		block = METAMORPH_CUDA_DEFAULT_BLOCK;
		grid = dim3((((*arr_end)[0] - (*arr_start)[0] + (block.x)) / block.x),
				(((*arr_end)[1] - (*arr_start)[1] + (block.y)) / block.y), 1);
		iters = (((*arr_end)[2] - (*arr_start)[2] + (block.z)) / block.z);
	} else {
		grid = dim3((*grid_size)[0], (*grid_size)[1], 1);
		block = dim3((*block_size)[0], (*block_size)[1], (*block_size)[2]);
		iters = (*grid_size)[2];
	}
	smem_len = block.x * block.y * block.z;
	if (event != NULL) {
		hipEventCreate(&(*event)[0]);
		hipEventRecord((*event)[0], 0);
	}
	//printf("CUDA Config: grid(%d, %d, %d) block(%d, %d, %d) iters %d\n", grid.x, grid.y, grid.z, block.x, block.y, block.z, iters);
	switch (type) {
	case a_db:

		kernel_reduction3<double><<<grid,block,smem_len*sizeof(double)>>>((double*)data, (*array_size)[0], (*array_size)[1], (*array_size)[2], (*arr_start)[0], (*arr_start)[1], (*arr_start)[2], (*arr_end)[0], (*arr_end)[1], (*arr_end)[2], iters, (double*)reduced_val, smem_len);
		break;

	case a_fl:
		kernel_reduction3<float><<<grid,block,smem_len*sizeof(float)>>>((float*)data, (*array_size)[0], (*array_size)[1], (*array_size)[2], (*arr_start)[0], (*arr_start)[1], (*arr_start)[2], (*arr_end)[0], (*arr_end)[1], (*arr_end)[2], iters, (float*)reduced_val, smem_len);
		break;

	case a_ul:
		kernel_reduction3<unsigned long long><<<grid,block,smem_len*sizeof(unsigned long long)>>>((unsigned long long*)data, (*array_size)[0], (*array_size)[1], (*array_size)[2], (*arr_start)[0], (*arr_start)[1], (*arr_start)[2], (*arr_end)[0], (*arr_end)[1], (*arr_end)[2], iters, (unsigned long long*)reduced_val, smem_len);
		break;

	case a_in:
		kernel_reduction3<int><<<grid,block,smem_len*sizeof(int)>>>((int*)data, (*array_size)[0], (*array_size)[1], (*array_size)[2], (*arr_start)[0], (*arr_start)[1], (*arr_start)[2], (*arr_end)[0], (*arr_end)[1], (*arr_end)[2], iters, (int*)reduced_val, smem_len);
		break;

	case a_ui:
		kernel_reduction3<unsigned int><<<grid,block,smem_len*sizeof(unsigned int)>>>((unsigned int*)data, (*array_size)[0], (*array_size)[1], (*array_size)[2], (*arr_start)[0], (*arr_start)[1], (*arr_start)[2], (*arr_end)[0], (*arr_end)[1], (*arr_end)[2], iters, (unsigned int*)reduced_val, smem_len);
		break;

	default:
		fprintf(stderr,
				"Error: Function 'cuda_reduce' not implemented for selected type!\n");
		break;

	}
	if (event != NULL) {
		hipEventCreate(&(*event)[1]);
		hipEventRecord((*event)[1], 0);
	}
	if (!async)
		ret = hipDeviceSynchronize();
	//TODO consider derailing for an explictly 2D/1D reduce..
	return (ret);
}

hipError_t cuda_transpose_face(size_t (*grid_size)[3],
		size_t (*block_size)[3], void *indata, void *outdata,
		size_t (*arr_dim_xy)[3], size_t (*tran_dim_xy)[3], meta_type_id type,
		int async, hipEvent_t ((*event)[2])) {
	hipError_t ret = hipSuccess;
	size_t smem_len;
	dim3 grid, block;
//	size_t smem_len = (*block_size)[0] * (*block_size)[1] * (*block_size)[2];
//TODO: Update to actually use user-provided grid/block once multi-element-per-thread
// scaling is added
//	dim3 grid = dim3((*grid_size)[0], (*grid_size)[1], 1);
//	dim3 block = dim3((*block_size)[0], (*block_size)[1], (*block_size)[2]);
	//FIXME: make this smart enough to rescale the threadblock (and thus shared memory - e.g. bank conflicts) w.r. double vs. float
	if (grid_size == NULL || block_size == NULL) {
		//FIXME: reconcile TILE_DIM/BLOCK_ROWS
		block = dim3(TRANSPOSE_TILE_DIM, TRANSPOSE_TILE_BLOCK_ROWS, 1);
		grid = dim3(((*tran_dim_xy)[0] + block.x - 1) / block.x,
				((*tran_dim_xy)[1] + block.y - 1) / block.y, 1);

	} else {
		grid = dim3((*grid_size)[0], (*grid_size)[1], 1);
		block = dim3((*block_size)[0], (*block_size)[1], (*block_size)[2]);
	}
	//The +1 here is to avoid bank conflicts with 32 floats or 16 doubles and is required by the kernel logic
	smem_len = (block.x + 1) * block.y * block.z;
	if (event != NULL) {
		hipEventCreate(&(*event)[0]);
		hipEventRecord((*event)[0], 0);
	}
	switch (type) {
	case a_db:
		kernel_transpose_2d<double><<<grid, block, smem_len*sizeof(double)>>>((double*)outdata, (double*)indata, (*arr_dim_xy)[0], (*arr_dim_xy)[1], (*tran_dim_xy)[0], (*tran_dim_xy)[1]);
		break;

	case a_fl:
		kernel_transpose_2d<float><<<grid, block, smem_len*sizeof(float)>>>((float*)outdata, (float*)indata, (*arr_dim_xy)[0], (*arr_dim_xy)[1], (*tran_dim_xy)[0], (*tran_dim_xy)[1]);
		break;

	case a_ul:
		kernel_transpose_2d<unsigned long><<<grid, block, smem_len*sizeof(unsigned long)>>>((unsigned long*)outdata, (unsigned long*)indata, (*arr_dim_xy)[0], (*arr_dim_xy)[1], (*tran_dim_xy)[0], (*tran_dim_xy)[1]);
		break;

	case a_in:
		kernel_transpose_2d<int><<<grid, block, smem_len*sizeof(int)>>>((int*)outdata, (int*)indata, (*arr_dim_xy)[0], (*arr_dim_xy)[1], (*tran_dim_xy)[0], (*tran_dim_xy)[1]);
		break;

	case a_ui:
		kernel_transpose_2d<unsigned int><<<grid, block, smem_len*sizeof(unsigned int)>>>((unsigned int*)outdata, (unsigned int*)indata, (*arr_dim_xy)[0], (*arr_dim_xy)[1], (*tran_dim_xy)[0], (*tran_dim_xy)[1]);
		break;

	default:
		fprintf(stderr,
				"Error: function 'cuda_transpose_face' not implemented for selected type!\n");
		break;
	}
	if (event != NULL) {
		hipEventCreate(&(*event)[1]);
		hipEventRecord((*event)[1], 0);
	}
	if (!async)
		ret = hipDeviceSynchronize();
	//TODO consider derailing for an explictly 2D/1D reduce..
	return (ret);
}

hipError_t cuda_pack_face(size_t (*grid_size)[3], size_t (*block_size)[3],
		void *packed_buf, void *buf, meta_face *face,
		int *remain_dim, meta_type_id type, int async,
		hipEvent_t ((*event_k1)[2]), hipEvent_t ((*event_c1)[2]),
		hipEvent_t ((*event_c2)[2]), hipEvent_t ((*event_c3)[2])) {
	hipError_t ret = hipSuccess;
	size_t smem_size;
	dim3 grid, block;
//TODO: Update to actually use user-provided grid/block once multi-element-per-thread
// scaling is added
//	dim3 grid = dim3((*grid_size)[0], (*grid_size)[1], 1);
//	dim3 block = dim3((*block_size)[0], (*block_size)[1], (*block_size)[2]);

	//copy required piece of the face struct into constant memory
	if (event_c1 != NULL) {
		hipEventCreate(&(*event_c1)[0]);
		hipEventRecord((*event_c1)[0], 0);
	}
	hipMemcpyToSymbol(HIP_SYMBOL(c_face_size), face->size, face->count * sizeof(int));
	if (event_c1 != NULL) {
		hipEventCreate(&(*event_c1)[1]);
		hipEventRecord((*event_c1)[1], 0);
	}

	if (event_c2 != NULL) {
		hipEventCreate(&(*event_c2)[0]);
		hipEventRecord((*event_c2)[0], 0);
	}
	hipMemcpyToSymbol(HIP_SYMBOL(c_face_stride), face->stride, face->count * sizeof(int));
	if (event_c2 != NULL) {
		hipEventCreate(&(*event_c2)[1]);
		hipEventRecord((*event_c2)[1], 0);
	}

	if (event_c3 != NULL) {
		hipEventCreate(&(*event_c3)[0]);
		hipEventRecord((*event_c3)[0], 0);
	}
	hipMemcpyToSymbol(HIP_SYMBOL(c_face_child_size), remain_dim,
			face->count * sizeof(int));
	if (event_c3 != NULL) {
		hipEventCreate(&(*event_c3)[1]);
		hipEventRecord((*event_c3)[1], 0);
	}
//TODO: Create a grid/block similar to Kaixi's look at mpi_wrap.c to figure out how size is computed
	if (event_k1 != NULL) {
		hipEventCreate(&(*event_k1)[0]);
		hipEventRecord((*event_k1)[0], 0);
	}
	//FIXME: specify a unique macro for each default blocksize
	if (grid_size == NULL || block_size == NULL) {
		block = dim3(256, 1, 1);
		grid = dim3(
				(face->size[0] * face->size[1] * face->size[2] + block.x - 1)
						/ block.x, 1, 1);
	} else {
		//This is a workaround for some non-determinism that was observed when allowing fully-arbitrary spec of grid/block
		if ((*block_size)[1] != 1 || (*block_size)[2] != 1
				|| (*grid_size)[1] != 1 || (*grid_size)[2])
			fprintf(stderr,
					"WARNING: Pack requires 1D block and grid, ignoring y/z params!\n");
		//block = dim3((*block_size)[0], (*block_size)[1], (*block_size)[2]);
		block = dim3((*block_size)[0], 1, 1);
		//grid = dim3((*grid_size)[0], (*grid_size)[1], 1);
		grid = dim3((*grid_size)[0], 1, 1);
	}
	smem_size = face->count * block.x * sizeof(int);
	switch (type) {
	case a_db:
		kernel_pack<double><<<grid, block, smem_size>>>((double *)packed_buf, (double *)buf, face->size[0]*face->size[1]*face->size[2], face->start, face->count);
		break;

	case a_fl:
		kernel_pack<float><<<grid, block, smem_size>>>((float *)packed_buf, (float *)buf, face->size[0]*face->size[1]*face->size[2], face->start, face->count);
		break;

	case a_ul:
		kernel_pack<unsigned long><<<grid, block, smem_size>>>((unsigned long *)packed_buf, (unsigned long *)buf, face->size[0]*face->size[1]*face->size[2], face->start, face->count);
		break;

	case a_in:
		kernel_pack<int><<<grid, block, smem_size>>>((int *)packed_buf, (int *)buf, face->size[0]*face->size[1]*face->size[2], face->start, face->count);
		break;

	case a_ui:
		kernel_pack<unsigned int><<<grid, block, smem_size>>>((unsigned int *)packed_buf, (unsigned int *)buf, face->size[0]*face->size[1]*face->size[2], face->start, face->count);
		break;

	default:
		fprintf(stderr,
				"Error: function 'cuda_pack_face' not implemented for selected type!\n");
		break;
	}
	if (event_k1 != NULL) {
		hipEventCreate(&(*event_k1)[1]);
		hipEventRecord((*event_k1)[1], 0);
	}
	if (!async)
		ret = hipDeviceSynchronize();
	//TODO consider derailing for an explictly 2D/1D reduce..
	return (ret);

}

hipError_t cuda_unpack_face(size_t (*grid_size)[3], size_t (*block_size)[3],
		void *packed_buf, void *buf, meta_face *face,
		int *remain_dim, meta_type_id type, int async,
		hipEvent_t ((*event_k1)[2]), hipEvent_t ((*event_c1)[2]),
		hipEvent_t ((*event_c2)[2]), hipEvent_t ((*event_c3)[2])) {
	hipError_t ret = hipSuccess;
	size_t smem_size;
	dim3 grid, block;
//TODO: Update to actually use user-provided grid/block once multi-element-per-thread
// scaling is added
//	dim3 grid = dim3((*grid_size)[0], (*grid_size)[1], 1);
//	dim3 block = dim3((*block_size)[0], (*block_size)[1], (*block_size)[2]);

	//copy required piece of the face struct into constant memory
	if (event_c1 != NULL) {
		hipEventCreate(&(*event_c1)[0]);
		hipEventRecord((*event_c1)[0], 0);
	}
	hipMemcpyToSymbol(HIP_SYMBOL(c_face_size), face->size, face->count * sizeof(int));
	if (event_c1 != NULL) {
		hipEventCreate(&(*event_c1)[1]);
		hipEventRecord((*event_c1)[1], 0);
	}

	if (event_c2 != NULL) {
		hipEventCreate(&(*event_c2)[0]);
		hipEventRecord((*event_c2)[0], 0);
	}
	hipMemcpyToSymbol(HIP_SYMBOL(c_face_stride), face->stride, face->count * sizeof(int));
	if (event_c2 != NULL) {
		hipEventCreate(&(*event_c2)[1]);
		hipEventRecord((*event_c2)[1], 0);
	}

	if (event_c3 != NULL) {
		hipEventCreate(&(*event_c3)[0]);
		hipEventRecord((*event_c3)[0], 0);
	}
	hipMemcpyToSymbol(HIP_SYMBOL(c_face_child_size), remain_dim,
			face->count * sizeof(int));
	if (event_c3 != NULL) {
		hipEventCreate(&(*event_c3)[1]);
		hipEventRecord((*event_c3)[1], 0);
	}
//TODO: Create a grid/block similar to Kaixi's look at mpi_wrap.c to figure out how size is computed
	if (event_k1 != NULL) {
		hipEventCreate(&(*event_k1)[0]);
		hipEventRecord((*event_k1)[0], 0);
	}
	//FIXME: specify a unique macro for each default blocksize
	if (grid_size == NULL || block_size == NULL) {
		block = dim3(256, 1, 1);
		grid = dim3(
				(face->size[0] * face->size[1] * face->size[2] + block.x - 1)
						/ block.x, 1, 1);
	} else {
		//This is a workaround for some non-determinism that was observed when allowing fully-arbitrary spec of grid/block
		if ((*block_size)[1] != 1 || (*block_size)[2] != 1
				|| (*grid_size)[1] != 1 || (*grid_size)[2])
			fprintf(stderr,
					"WARNING: Unpack requires 1D block and grid, ignoring y/z params!\n");
		//block = dim3((*block_size)[0], (*block_size)[1], (*block_size)[2]);
		block = dim3((*block_size)[0], 1, 1);
		//grid = dim3((*grid_size)[0], (*grid_size)[1], 1);
		grid = dim3((*grid_size)[0], 1, 1);
	}
	smem_size = face->count * block.x * sizeof(int);
	switch (type) {
	case a_db:
		kernel_unpack<double><<<grid, block, smem_size>>>((double *)packed_buf, (double *)buf, face->size[0]*face->size[1]*face->size[2], face->start, face->count);
		break;

	case a_fl:
		kernel_unpack<float><<<grid, block, smem_size>>>((float *)packed_buf, (float *)buf, face->size[0]*face->size[1]*face->size[2], face->start, face->count);
		break;

	case a_ul:
		kernel_unpack<unsigned long><<<grid, block, smem_size>>>((unsigned long *)packed_buf, (unsigned long *)buf, face->size[0]*face->size[1]*face->size[2], face->start, face->count);
		break;

	case a_in:
		kernel_unpack<int><<<grid, block, smem_size>>>((int *)packed_buf, (int *)buf, face->size[0]*face->size[1]*face->size[2], face->start, face->count);
		break;

	case a_ui:
		kernel_unpack<unsigned int><<<grid, block, smem_size>>>((unsigned int *)packed_buf, (unsigned int *)buf, face->size[0]*face->size[1]*face->size[2], face->start, face->count);
		break;

	default:
		fprintf(stderr,
				"Error: function 'cuda_unpack_face' not implemented for selected type!\n");
		break;
	}
	if (event_k1 != NULL) {
		hipEventCreate(&(*event_k1)[1]);
		hipEventRecord((*event_k1)[1], 0);
	}
	if (!async)
		ret = hipDeviceSynchronize();
	//TODO consider derailing for an explictly 2D/1D reduce..
	return (ret);

}

hipError_t cuda_stencil_3d7p(size_t (*grid_size)[3], size_t (*block_size)[3],
		void * indata, void * outdata, size_t (*array_size)[3],
		size_t (*arr_start)[3], size_t (*arr_end)[3], meta_type_id type,
		int async, hipEvent_t ((*event)[2])) {
	hipError_t ret = hipSuccess;
	size_t smem_len;
	dim3 grid;
	dim3 block;
	int iters;
	//Allow for auto-selected grid/block size if either is not specified
	if (grid_size == NULL || block_size == NULL) {
		//block = METAMORPH_CUDA_DEFAULT_BLOCK;
		block = dim3(64, 4, 1);
		//do not subtract 1 from blocx for the case when end == start
		grid = dim3((((*arr_end)[0] - (*arr_start)[0] + (block.x)) / block.x),
				(((*arr_end)[1] - (*arr_start)[1] + (block.y)) / block.y), 1);
		iters = (((*arr_end)[2] - (*arr_start)[2] + (block.z)) / block.z);
	} else {
		grid = dim3((*grid_size)[0], (*grid_size)[1], 1);
		block = dim3((*block_size)[0], (*block_size)[1], (*block_size)[2]);
		iters = (int) (*grid_size)[2];
	}
	smem_len = (block.x + 2) * (block.y + 2) * block.z;
	//smem_len = 0;

	if (event != NULL) {
		hipEventCreate(&(*event)[0]);
		hipEventRecord((*event)[0], 0);
	}

	switch (type) {
	case a_db:
		kernel_stencil_3d7p<double><<<grid,block,smem_len*sizeof(double)>>>((double*)indata, (double*)outdata, (*array_size)[0], (*array_size)[1], (*array_size)[2], (*arr_start)[0], (*arr_start)[1], (*arr_start)[2], (*arr_end)[0], (*arr_end)[1], (*arr_end)[2], iters, smem_len);
		break;

	case a_fl:
		kernel_stencil_3d7p<float><<<grid,block,smem_len*sizeof(float)>>>((float*)indata, (float*)outdata, (*array_size)[0], (*array_size)[1], (*array_size)[2], (*arr_start)[0], (*arr_start)[1], (*arr_start)[2], (*arr_end)[0], (*arr_end)[1], (*arr_end)[2], iters, smem_len);
		break;

	case a_ul:
		kernel_stencil_3d7p<unsigned long long><<<grid,block,smem_len*sizeof(unsigned long long)>>>((unsigned long long*)indata, (unsigned long long*)outdata, (*array_size)[0], (*array_size)[1], (*array_size)[2], (*arr_start)[0], (*arr_start)[1], (*arr_start)[2], (*arr_end)[0], (*arr_end)[1], (*arr_end)[2], iters, smem_len);
		break;

	case a_in:
		kernel_stencil_3d7p<int><<<grid,block,smem_len*sizeof(int)>>>((int*)indata, (int*)outdata, (*array_size)[0], (*array_size)[1], (*array_size)[2], (*arr_start)[0], (*arr_start)[1], (*arr_start)[2], (*arr_end)[0], (*arr_end)[1], (*arr_end)[2], iters, smem_len);
		break;

	case a_ui:
		kernel_stencil_3d7p<unsigned int><<<grid,block,smem_len*sizeof(unsigned int)>>>((unsigned int*)indata, (unsigned int*)outdata, (*array_size)[0], (*array_size)[1], (*array_size)[2], (*arr_start)[0], (*arr_start)[1], (*arr_start)[2], (*arr_end)[0], (*arr_end)[1], (*arr_end)[2], iters, smem_len);
		break;

	default:
		fprintf(stderr,
				"Error: Function 'cuda_stencil_3d7p' not implemented for selected type!\n");
		break;
	}

	if (event != NULL) {
		hipEventCreate(&(*event)[1]);
		hipEventRecord((*event)[1], 0);
	}
	if (!async)
		ret = hipDeviceSynchronize();
	return (ret);

}
